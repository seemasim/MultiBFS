#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <fstream>
#include <algorithm>
#include <string>
#include  <thrust/equal.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>



void printMatrix(int m, int n, const float*A, int lda, const char* name)
{
	for(int row = 0 ; row < m ; row++){
		for(int col = 0 ; col < n ; col++){
			float Areg = A[row + col*lda];
			printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
		}
	}
}


struct my_policy : thrust::device_execution_policy<my_policy> {};

int main(int argc, char*argv[])
{

	hipsparseMatDescr_t descrA = NULL;
	hipsparseMatDescr_t descrB = NULL;
	hipsparseMatDescr_t descrC = NULL;

	hipblasStatus_t cublasStat = HIPBLAS_STATUS_SUCCESS;
	hipsparseStatus_t cusparseStat = HIPSPARSE_STATUS_SUCCESS;
	hipError_t cudaStat1 = hipSuccess;
	hipError_t cudaStat2 = hipSuccess;
	hipError_t cudaStat3 = hipSuccess;
	
	const int n = 4847571;
	const size_t nnzA = 68993773 + n;
	//const int n = 7;
	//const size_t nnzA = 19;
	size_t nnzB = n;
	
	const int *csrRowPtrA = (const int*)malloc(sizeof(const int)*(n+1));
	const int *csrColIndA = (const int*)malloc(sizeof(const int)*(nnzA));
	const float csrValA[nnzA] = { 0.0 };


	int *csrRowPtrB = (int *)malloc(sizeof(int)*(n+1));
	int *csrColIndB = (int *)malloc(sizeof(int)*(nnzB));
	float csrValB[nnzB];

	int i;
	for(i=0;i<n;i++){
		*(csrColIndB + i) = i;
		*(csrRowPtrB + i) = i;
		csrValB[i] = 1.0;
	}
	*(csrRowPtrB + i) = i;


	{
		//std::ifstream file("value_pokec.txt");
		std::ifstream file("value.txt");
		std::string str;
		int i = 0;
		while (std::getline(file, str)) {
			float &ptr = const_cast <float &>(csrValA[i]); 
			ptr = (const float)atoi(str.c_str()); 
			i++;
		}
	}

	{
		//std::ifstream file("indices_pokec.txt");
		std::ifstream file("indices.txt");
		int i = 0;
		std::string str;
		while (std::getline(file, str)) {
			int &ptr = const_cast <int &>(csrColIndA[i]);
			ptr = (const int)atoi(str.c_str());
			i++;
		}
	}

	{
		//std::ifstream file("indptr_pokec.txt");
		std::ifstream file("indptr.txt");
		int i = 0;
		std::string str;
		while (std::getline(file, str)) {
			int &ptr = const_cast <int &>(csrRowPtrA[i]);
			ptr = (const int)atoi(str.c_str());
			i++;
		}
	}



	int cscColPtrA[n+1];
	int cscRowIndA[nnzA];
	float cscValA[nnzA];

	int *d_csrRowPtrA = NULL;
	int *d_csrColIndA = NULL;
	float *d_csrValA = NULL;

	int *d_csrRowPtrB = NULL;
	int *d_csrColIndB = NULL;
	float *d_csrValB = NULL;

	int *d_cscColPtrA = NULL;
	int *d_cscRowIndA = NULL;
	float *d_cscValA = NULL;


	/* step 2: configuration of matrix A */
	cusparseStat = hipsparseCreateMatDescr(&descrA);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

	hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

	/* configuration of matrix B */

	cusparseStat = hipsparseCreateMatDescr(&descrB);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);   

	hipsparseSetMatIndexBase(descrB,HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL );

	/* step 3: copy A,B and x0 to device */
	cudaStat1 = hipMalloc ((void**)&d_csrRowPtrA, sizeof(int) * (n+1) );
	cudaStat2 = hipMalloc ((void**)&d_csrColIndA, sizeof(int) * nnzA );
	cudaStat3 = hipMalloc ((void**)&d_csrValA   , sizeof(float) * nnzA );

	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);

	cudaStat1 = hipMalloc ((void**)&d_cscColPtrA, sizeof(int) * (n+1) );
	cudaStat2 = hipMalloc ((void**)&d_cscRowIndA, sizeof(int) * nnzA );
	cudaStat3 = hipMalloc ((void**)&d_cscValA   , sizeof(float) * nnzA );

	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);

	cudaStat1 = hipMalloc ((void**)&d_csrRowPtrB, sizeof(int) * (n+1) );
	thrust::device_ptr<int> v2RowPtrB(d_csrRowPtrB);
	cudaStat2 = hipMalloc ((void**)&d_csrColIndB, sizeof(int) * n );
	thrust::device_ptr<int> v1ColIndB(d_csrColIndB);
	cudaStat3 = hipMalloc ((void**)&d_csrValB   , sizeof(float) * n );

	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);


	cudaStat1 = hipMemcpy(d_csrRowPtrA, csrRowPtrA, sizeof(int) * (n+1)   , hipMemcpyHostToDevice);
	cudaStat2 = hipMemcpy(d_csrColIndA, csrColIndA, sizeof(int) * nnzA    , hipMemcpyHostToDevice);
	cudaStat3 = hipMemcpy(d_csrValA   , csrValA   , sizeof(float) * nnzA , hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);

	cudaStat1 = hipMemcpy(d_csrRowPtrB, csrRowPtrB, sizeof(int) * (n+1)   , hipMemcpyHostToDevice);

	cudaStat2 = hipMemcpy(d_csrColIndB, csrColIndB, sizeof(int) * n    , hipMemcpyHostToDevice);
	cudaStat3 = hipMemcpy(d_csrValB   , csrValB   , sizeof(float) * n , hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);

	hipblasHandle_t cublasH = NULL;
	hipsparseHandle_t cusparseH = NULL;
	hipStream_t stream = NULL;

	/* step 1: create cublas/cusparse handle, bind a stream */
	cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	assert(hipSuccess == cudaStat1);

	cublasStat = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublasStat);

	cublasStat = hipblasSetStream(cublasH, stream);
	assert(HIPBLAS_STATUS_SUCCESS == cublasStat);

	cusparseStat = hipsparseCreate(&cusparseH);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

	cusparseStat = hipsparseSetStream(cusparseH, stream);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

	cusparseStat = hipsparseScsr2csc(cusparseH, 
			n, 
			n, 
			nnzA,
			d_csrValA, 
			d_csrRowPtrA,
			d_csrColIndA, 
			d_cscValA, 
			d_cscRowIndA,
			d_cscColPtrA, 
			HIPSPARSE_ACTION_NUMERIC, 
			HIPSPARSE_INDEX_BASE_ZERO);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);


	cudaStat1 = hipMemcpy(cscValA, d_cscValA, sizeof(float) * nnzA, hipMemcpyDeviceToHost);
	cudaStat2 = hipMemcpy(cscRowIndA, d_cscRowIndA, sizeof(int) * nnzA, hipMemcpyDeviceToHost);
	cudaStat3 = hipMemcpy(cscColPtrA, d_cscColPtrA, sizeof(int) * (n+1), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);

	 

	delete[] csrRowPtrA;
	delete [] csrColIndA;
	

	//Configure matrix C
	cusparseStat = hipsparseCreateMatDescr(&descrC);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);   

	cusparseStat = hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ZERO);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);   

	cusparseStat = hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL );
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);   

	int iteration = 0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float time_in_ms = 0;

	while(1){ 
		printf("iteration==%d\n",iteration );

		int *d_csrRowPtrC = NULL;
		int *d_csrColIndC = NULL;
		float *d_csrValC = NULL; //x0

		
		int baseC,nnzC;

		// nnzTotalDevHostPtr points to host memory
		int *nnzTotalDevHostPtr = (int*)&nnzC;

		cusparseStat = hipsparseSetPointerMode(cusparseH, HIPSPARSE_POINTER_MODE_HOST);
		assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);


		cudaStat1 = hipMalloc((void**)&d_csrRowPtrC, sizeof(int)*(n+1));
		thrust::device_ptr<int> v2RowPtrC(d_csrRowPtrC);
	
		assert(hipSuccess == cudaStat1);
			

		//d_cscColPtrA is used because we want the transpose of matrix A to be used
		float time_nnzC;
		hipEventRecord(start,stream);
		cusparseStat = hipsparseXcsrgemmNnz(cusparseH,HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, n,
				descrA, nnzA, d_cscColPtrA, d_cscRowIndA,
				descrB, nnzB, d_csrRowPtrB, d_csrColIndB,
				descrC, d_csrRowPtrC, nnzTotalDevHostPtr);


		assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);  
		hipEventRecord(stop,stream);

		if (NULL != nnzTotalDevHostPtr){
			nnzC = *nnzTotalDevHostPtr;
		}else{
			hipMemcpy(&nnzC, d_csrRowPtrC+n, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(&baseC, d_csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
			nnzC -= baseC;
		}
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_nnzC, start, stop);

		hipMalloc((void**)&d_csrColIndC, sizeof(int)*nnzC);
		thrust::device_ptr<int> v1ColIndC(d_csrColIndC);
		hipMalloc((void**)&d_csrValC, sizeof(float)*nnzC);

		printf("nnzC=%d\n",nnzC);

	
		float time_mm;
		hipEventRecord(start, stream);
		cusparseStat = hipsparseScsrgemm(cusparseH, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, n,
				descrA, nnzA,
				d_cscValA, d_cscColPtrA, d_cscRowIndA,
				descrB, nnzB,
				d_csrValB, d_csrRowPtrB, d_csrColIndB,
				descrC,
				d_csrValC, d_csrRowPtrC, d_csrColIndC);
		assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
		hipEventRecord(stop, stream);

		

		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_mm, start, stop);

		

		thrust::device_ptr<int> v2RowPtrB(d_csrRowPtrB);
		thrust::device_ptr<int> v1ColIndB(d_csrColIndB);
		bool flag1 = false;
		bool flag2 = false;
		//size_t N = nnzB;
		my_policy exec;

		//compare matrix B with matrix C because matrix B had previous A*B and C has new A*B
		float time_eqlComparison;
		if(iteration != 0){			
			
			hipEventRecord(start, stream);printf("recording...\n");
			flag1 = thrust::equal(exec, v1ColIndB, v1ColIndB + nnzB, v1ColIndC);			
			flag2 = thrust::equal(exec, v2RowPtrB, v2RowPtrB + n, v2RowPtrC);
			hipEventRecord(stop, stream);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&time_eqlComparison, start, stop);
			if(flag1 == true && flag2 == true){
				printf("**CONVERGED** the previous two iteration are same\n");
				if (d_csrRowPtrC  ) hipFree(d_csrRowPtrC);
				if (d_csrColIndC  ) hipFree(d_csrColIndC);
				if (d_csrValC     ) hipFree(d_csrValC);
				if (descrC        ) hipsparseDestroyMatDescr(descrC);
				break;
			}
		}
		nnzB = nnzC;


		if (d_csrRowPtrB  ) hipFree(d_csrRowPtrB);
		if (d_csrColIndB  ) hipFree(d_csrColIndB);
		if (d_csrValB     ) hipFree(d_csrValB);


		cudaStat1 = hipMalloc ((void**)&d_csrRowPtrB, sizeof(int) * (n+1) );		
		cudaStat2 = hipMalloc ((void**)&d_csrColIndB, sizeof(int) * nnzB );		
		cudaStat3 = hipMalloc ((void**)&d_csrValB   , sizeof(float) * nnzB );
		

		assert(hipSuccess == cudaStat1);
		assert(hipSuccess == cudaStat2);
		assert(hipSuccess == cudaStat3);


		cudaStat1 = hipMemcpy(d_csrRowPtrB, d_csrRowPtrC, sizeof(int) * (n+1), hipMemcpyDeviceToDevice);
		cudaStat2 = hipMemcpy(d_csrColIndB, d_csrColIndC, sizeof(int) * nnzB    , hipMemcpyDeviceToDevice);
		cudaStat3 = hipMemcpy(d_csrValB   , d_csrValC   , sizeof(float) * nnzB , hipMemcpyDeviceToDevice);
		assert(hipSuccess == cudaStat1);
		assert(hipSuccess == cudaStat2);
		assert(hipSuccess == cudaStat3);

		if (d_csrRowPtrC  ) hipFree(d_csrRowPtrC);
		if (d_csrColIndC  ) hipFree(d_csrColIndC);
		if (d_csrValC     ) hipFree(d_csrValC);

		iteration++;
		float x=time_nnzC + time_mm ;
		printf("time elapsed in this iteration=%f\n",x);
		time_in_ms += x;
		
		} //end while

	printf("total time==%f\n",time_in_ms);
		//if iteration is 1 then store the result matrix to X for the 
		//remaining iterations compare the two matrices.
		hipEventDestroy(start);
		hipEventDestroy(stop);

		if (cublasH       ) hipblasDestroy(cublasH);
		if (cusparseH     ) hipsparseDestroy(cusparseH);
		if (stream        ) hipStreamDestroy(stream);
		/* free resources */
		if (d_csrRowPtrA  ) hipFree(d_csrRowPtrA);
		if (d_csrColIndA  ) hipFree(d_csrColIndA);
		if (d_csrValA     ) hipFree(d_csrValA);
		if (descrA        ) hipsparseDestroyMatDescr(descrA);

		if (d_csrRowPtrB  ) hipFree(d_csrRowPtrB);
		if (d_csrColIndB  ) hipFree(d_csrColIndB);
		if (d_csrValB     ) hipFree(d_csrValB);
		if (descrB        ) hipsparseDestroyMatDescr(descrB);


		printf("9\n");
		hipDeviceReset();

		return 0;
	}
